
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

__global__ void vecAdd(float* h_a, float* h_b, float* h_c, int n)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;

	//check if it is in bound
	if(id<n)
		h_c[id] = h_a[id]+ h_b[id];

}

int main(int argc, char* argv[])
{
	//size of vectors
	int n= 1000;
	
	float *h_a;//ip
	float *h_b;//ip
	float *h_c;//op
	
	float *d_a;//ip
	float *d_b;//ip
	float *d_c;//op

	int size = n * sizeof(float);		
	
	//allocating memory on host
	h_a = (float*)malloc(size);
	h_b = (float*)malloc(size);
	h_c = (float*)malloc(size);	
	
	//allocating memory for each vector on GPU
	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_c, size);
	
	//initialize vectors on host
	int i;
	for(i = 0; i<n; i++)
	{
		h_a[i] = sin(i)*sin(i);
		h_b[i] = cos(i)*cos(i);
	}

	/*printf("h_a: \n");
	for(i=0; i<n; i++)
		printf("%.1f\n", h_a[i]);
	printf("\n");

	printf("h_b: \n");
	for(i=0; i<n; i++)
		printf("%.1f\n", h_b[i]);
	printf("\n");
	*/

	//copy host vectors to device
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
	
	int threadPerBlocks, blockCount;
	
	//block size
	threadPerBlocks = 1024;
	
	//grid size
	blockCount = (int)ceil((float)n/threadPerBlocks);

	//executing kernel 
	vecAdd<<<threadPerBlocks, blockCount>>>(d_a, d_b, d_c, n);
	
	//copy array back to host
	hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
	
	float sum = 0;
	for(i=0; i<n; i++)
		sum += h_c[i];
	printf("Final result is: %f\n", sum/n);
	
	//release device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	//releasing host memory
	free(h_a);
	free(h_b);
	free(h_c);

	return 0;
	
	
}
